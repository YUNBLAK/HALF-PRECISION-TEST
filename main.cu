#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_fp16.h>
#include <time.h>

int *garr01, *garr02, *garr03;
int *arr01, *arr02, *arr03;
__half *bigArray;

// nvcc main.cu -o main -gencode arch=compute_75,code=[sm_75,compute_75]

__device__ __half2 Taylor_exponential_fp16(int n, __half2 x) { 
    __half2 exp_sum = __int2half2_rd(1);  
    __half2 initone = __int2half2_rd(1);   
    for (long int i = n - 1; i > 0; --i){
        //exp_sum = __hadd2(initone, __hdiv2((__hmul2(x, exp_sum)), __int2half2_rd(i)));
        exp_sum = __hcmadd(hmul2(x, exp_sum), hrcp(__int2half2_rd(i)), initone);
    }    
    return exp_sum; 
}

__device__ float Taylor_exponential_fp32(int n, float x) { 
    float exp_sum = 1;     
    for (long int i = n - 1; i > 0; --i ) 
        exp_sum = 1 + x * exp_sum / i;    
    return exp_sum; 
}

__device__ double Taylor_exponential_fp64(int n, double x) { 
    double exp_sum = 1;     
    for (long int i = n - 1; i > 0; --i ) 
        exp_sum = 1 + x * exp_sum / i;    
    return exp_sum; 
}

__global__ void gpuCal(int N, float x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("[] %d\t %d\t %d\t %d\t\n", i, blockIdx.x, blockDim.x, threadIdx.x);
    
    #ifdef FP16
        printf ("[FP Float16 used]\n");
        __half xx = Taylor_exponential_fp16(N, __float2half(x));
        printf("\ne^x = %g\n",__half2float(xx));
    #else 
        printf ("[FP Float32 used]\n");
        float xx = Taylor_exponential_fp32(N, x);
        printf("\ne^x = %g\n",xx);
    #endif
}


int main(int argc, char *argv[]) 
{
    float x = 0.5;
    int taylor_N = 20000;
    int big_N = 1;

    int n = 1;
    int nBytes = n * sizeof(int);
    int block_size = 1;
    int block_no = (n + block_size - 1)/block_size;

    clock_t start, end;
    double cpu_time_used;

    char *a = argv[1];

    if (argc == 1 ){
        printf("Missing argument 'n', defaults to %ld.\n", taylor_N);
    }
    
    else {
        char *a = argv[1];
        taylor_N = atoi(a);
    }

    #ifdef FP16
        printf ("[FP Float16 used]\n");
    #else
        printf ("[FP Float32 used]\n");
    #endif
    
    printf("value of n = %ld and x = %g ", taylor_N, x);

    start = clock();
    gpuCal<<<block_no, block_size>>>(taylor_N, x);
    hipDeviceSynchronize();
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf ("CPU time: %g seconds\n", cpu_time_used);

    return 0;
}