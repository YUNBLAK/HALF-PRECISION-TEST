#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include <hip/hip_fp16.h>

int *garr01, *garr02, *garr03;
int *arr01, *arr02, *arr03;

__global__ void vecAdd(int *A, int *B, int *C, int N)
{
    half x = __float2half(0.0);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        C[i] = A[i] + B[i];
    
}

int main() 
{
    half * dh0, *dh1, *dh2;
    int n = 100;
    int nBytes = n * sizeof(int);
    int block_size = 32;
    int block_no = (n + block_size - 1)/block_size;

    arr01 = (int *)malloc(nBytes);
    arr02 = (int *)malloc(nBytes);
    arr03 = (int *)malloc(nBytes);

    for(int i = 0; i<n; i++){
        arr01[i] = i;
        arr02[i] = i*i;
    }

    printf("Allocating device memory on host\n");
    hipMalloc((void **)&garr01, n*sizeof(int));
    hipMalloc((void **)&garr02, n*sizeof(int));
    hipMalloc((void **)&garr03, n*sizeof(int));
    
    printf("Copying to device\n");
    hipMemcpy(garr01, arr01, n *sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(garr02, arr02, n *sizeof(int), hipMemcpyHostToDevice);
    //cudaMemcpy(garr03, arr03, n *sizeof(int), cudaMemcpyHostToDevice);
    
    printf("Doing GPU Vector\n");
    vecAdd<<<block_no, block_size>>>(garr01, garr02, garr03, n);

    printf("SYNC\n");
    hipDeviceSynchronize();

    // vecAddOne_h(host_A, host_C1, n);
    hipMemcpy(arr03, garr03, n*sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i =0;i<n; i++){
        printf("%d\n", arr03[i]);
    }
    
    hipFree(garr01);
    hipFree(garr02);
    hipFree(garr03);

    return 0;
}
